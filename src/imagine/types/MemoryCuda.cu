#include "imagine/types/MemoryCuda.hpp"

#include <hip/hip_runtime.h>

namespace imagine {

// CUDA HELPER
namespace cuda {

void* memcpyHostToDevice(void* dest, const void* src, std::size_t count)
{
    CUDA_DEBUG( hipMemcpy(dest, src, count, hipMemcpyHostToDevice) );
    return dest;
}

void* memcpyDeviceToHost(void* dest, const void* src, std::size_t count)
{
    CUDA_DEBUG( hipMemcpy(dest, src, count, hipMemcpyDeviceToHost) );
    return dest;
}

void* memcpyDeviceToDevice(void* dest, const void* src, std::size_t count)
{
    CUDA_DEBUG( hipMemcpy(dest, src, count, hipMemcpyDeviceToDevice) );
    return dest;
}

void* memcpyHostToHost(void* dest, const void* src, std::size_t count)
{
    CUDA_DEBUG( hipMemcpy(dest, src, count, hipMemcpyHostToHost) );
    return dest;
}

} // namespace cuda

void* VRAM_CUDA::alloc(size_t N)
{
    void* ret;
    CUDA_DEBUG( hipMalloc(&ret, N) );
    return ret;
}

void* VRAM_CUDA::realloc(void* mem, size_t N)
{
    void* ret;
    CUDA_DEBUG( hipMalloc(&ret, N) );
    // what if N smaller then old memory?
    // hipMemcpy(&ret, mem, N, hipMemcpyDeviceToDevice);
    CUDA_DEBUG( hipFree(mem) );
    return ret;
}

void VRAM_CUDA::free(void* mem)
{
    CUDA_DEBUG( hipFree(mem) );
}

// RAM CUDA
void* RAM_CUDA::alloc(size_t N)
{
    void* ret;
    CUDA_DEBUG( hipHostMalloc(&ret, N) );
    return ret;
}

void* RAM_CUDA::realloc(void* mem, size_t N)
{
    void* ret;
    CUDA_DEBUG( hipHostMalloc(&ret, N) );
    CUDA_DEBUG( hipHostFree(mem) );
    return ret;
}

void RAM_CUDA::free(void* mem)
{
    CUDA_DEBUG( hipHostFree(mem) );
}

} // namespace mamcl