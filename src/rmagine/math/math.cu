#include "hip/hip_runtime.h"
#include "rmagine/math/math.cuh"
#include <rmagine/math/math.h>
#include <rmagine/math/types.h>

namespace rmagine 
{

__global__ void multNxN_kernel(
    const Quaternion* A,
    const Quaternion* B,
    Quaternion* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] * B[id];
    }
}

void multNxN(
    const Memory<Quaternion, VRAM_CUDA>& A,
    const Memory<Quaternion, VRAM_CUDA>& B,
    Memory<Quaternion, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    multNxN_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
}

Memory<Quaternion, VRAM_CUDA> multNxN(
    const Memory<Quaternion, VRAM_CUDA>& A, 
    const Memory<Quaternion, VRAM_CUDA>& B)
{
    Memory<Quaternion, VRAM_CUDA> C(A.size());
    // mult
    multNxN(A, B, C);
    return C;
}

__global__ void multNxN_kernel(
    const Quaternion* A,
    const Vector* b,
    Vector* c,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        c[id] = A[id] * b[id];
    }
}

void multNxN(
    const Memory<Quaternion, VRAM_CUDA>& A,
    const Memory<Vector, VRAM_CUDA>& b, 
    Memory<Vector, VRAM_CUDA>& c)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    multNxN_kernel<<<gridSize, blockSize>>>(A.raw(), b.raw(), c.raw(), A.size());
}

Memory<Vector, VRAM_CUDA> multNxN(
    const Memory<Quaternion, VRAM_CUDA>& A,
    const Memory<Vector, VRAM_CUDA>& b)
{
    Memory<Vector, VRAM_CUDA> c(A.size());
    multNxN(A, b, c);
    return c;
}

__global__ void multNxN_kernel(
    const Transform* T,
    const Vector* x,
    Vector* c,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        c[id] = T[id] * x[id];
    }
}

void multNxN(
    const Memory<Transform, VRAM_CUDA>& T,
    const Memory<Vector, VRAM_CUDA>& x,
    Memory<Vector, VRAM_CUDA>& c)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (T.size() + blockSize - 1) / blockSize;
    multNxN_kernel<<<gridSize, blockSize>>>(T.raw(), x.raw(), c.raw(), T.size());
}

Memory<Vector, VRAM_CUDA> multNxN(
    const Memory<Transform, VRAM_CUDA>& T,
    const Memory<Vector, VRAM_CUDA>& x)
{
    Memory<Vector, VRAM_CUDA> c(T.size());
    multNxN(T,x,c);
    return c;
}

__global__ void multNxN_kernel(
    const Matrix3x3* M,
    const Vector* x,
    Vector* c,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        c[id] = M[id] * x[id];
    }
}

void multNxN(
    const Memory<Matrix3x3, VRAM_CUDA>& M,
    const Memory<Vector, VRAM_CUDA>& x,
    Memory<Vector, VRAM_CUDA>& c)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M.size() + blockSize - 1) / blockSize;
    multNxN_kernel<<<gridSize, blockSize>>>(M.raw(), x.raw(), c.raw(), M.size());
}

Memory<Vector, VRAM_CUDA> multNxN(
    const Memory<Matrix3x3, VRAM_CUDA>& M,
    const Memory<Vector, VRAM_CUDA>& x)
{
    Memory<Vector, VRAM_CUDA> c(M.size());
    multNxN(M, x, c);
    return c;
}

} // namespace rmagine