#include "hip/hip_runtime.h"
#include "rmagine/math/math.cuh"
#include "rmagine/math/math.h"
#include "rmagine/math/types.h"

namespace rmagine 
{

////////
// Generic Kernel
///

template<typename In1T, typename In2T, typename ResT>
__global__ void multNxN_kernel(
    const In1T* A,
    const In2T* B,
    ResT* C,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] * B[id];
    }
}

template<typename In1T, typename In2T, typename ResT>
__global__ void multNx1_kernel(
    const In1T* A,
    const In2T* b,
    ResT* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] * b[0];
    }
}

template<typename In1T, typename In2T, typename ResT>
__global__ void mult1xN_kernel(
    const In1T* a,
    const In2T* B,
    ResT* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = a[0] * B[id];
    }
}


template<typename In1T, typename In2T, typename ResT>
__global__ void addNxN_kernel(
    const In1T* A,
    const In2T* B,
    ResT* C,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] + B[id];
    }
}

template<typename In1T, typename In2T, typename ResT>
__global__ void subNxN_kernel(
    const In1T* A,
    const In2T* B,
    ResT* C,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] - B[id];
    }
}

template<typename In1T, typename In2T, typename ResT>
__global__ void subNx1_kernel(
    const In1T* A,
    const In2T* b,
    ResT* C,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] - b[0];
    }
}


template<typename T>
__global__ void transpose_kernel(
    const T* A,
    T* B,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        B[id] = A[id].transpose();
    }
}

template<typename T>
__global__ void transposeInplace_kernel(
    T* A,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        A[id].transposeInplace();
    }
}


template<typename T>
__global__ void invert_kernel(
    const T* A,
    T* B,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        B[id] = A[id].inv();
    }
}

template<typename In1T, typename In2T, typename ResT>
__global__ void divNxN_kernel(
    const In1T* A, 
    const In2T* B,
    ResT* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] / B[id];
    }
}

template<typename ConvT, typename In1T, typename In2T, typename ResT>
__global__ void divNxN_conv_kernel(
    const In1T* A, 
    const In2T* B,
    ResT* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] / static_cast<ConvT>(B[id]);
    }
}

template<typename ConvT>
__global__ void divNxNIgnoreZeros_conv_kernel(
    const Vector* A, 
    const unsigned int* B,
    Vector* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        if(B[id] > 0)
        {
            C[id] = A[id] / static_cast<ConvT>(B[id]);
        } else {
            C[id].setZeros();
        }
    }
}

template<typename ConvT>
__global__ void divNxNIgnoreZeros_conv_kernel(
    const Matrix3x3* A,
    const unsigned int* B,
    Matrix3x3* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        if(B[id] > 0)
        {
            C[id] = A[id] / static_cast<ConvT>(B[id]);
        } else {
            C[id].setZeros();
        }
    }
}

template<typename ConvT>
__global__ void divNxNIgnoreZeros_conv_kernel(
    const float* A, 
    const unsigned int* B,
    float* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        if(B[id] > 0)
        {
            C[id] = A[id] / static_cast<ConvT>(B[id]);
        } else {
            C[id] = 0.0;
        }
    }
}

__global__ void divNxNInplace_kernel(
    Vector* A, 
    const float* B,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        A[id] /= B[id];
    }
}

__global__ void divNxNInplace_kernel(
    Matrix3x3* A, 
    const unsigned int* B,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        A[id] /= static_cast<float>(B[id]);
    }
}

template<typename T>
__global__ void divNx1Inplace_kernel(
    T* A,
    unsigned int b,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        A[id] /= static_cast<float>(b);
    }
}

__global__ void convert_kernel(const uint8_t* from, float* to, unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        to[id] = static_cast<float>(from[id]);
    }
}

__global__
void convert_kernel(const bool* from, unsigned int* to, unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        to[id] = static_cast<unsigned int>(from[id]);
    }
}

__global__
void convert_kernel(const unsigned int* from, bool* to, unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        to[id] = (from[id] > 0);
    }
}

__global__ void pack_kernel(
    const Matrix3x3* R, 
    const Vector* t, // Vector3d / Vector3f
    Transform* T,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        T[id].R.set(R[id]);
        T[id].t = t[id];
    }
}

__global__ void pack_kernel(
    const Quaternion* R, 
    const Vector* t, // Vector3d / Vector3f
    Transform* T,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        T[id].R = R[id];
        T[id].t = t[id];
    }
}

__global__ void covParts_kernel(
    const Vector* a, 
    const Vector* b,
    Matrix3x3* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id](0,0) = a[id].x * b[id].x;
        C[id](1,0) = a[id].x * b[id].y;
        C[id](2,0) = a[id].x * b[id].z;
        C[id](0,1) = a[id].y * b[id].x;
        C[id](1,1) = a[id].y * b[id].y;
        C[id](2,1) = a[id].y * b[id].z;
        C[id](0,2) = a[id].z * b[id].x;
        C[id](1,2) = a[id].z * b[id].y;
        C[id](2,2) = a[id].z * b[id].z;
    }
}

__global__ void covParts_kernel(
    const Vector* a, 
    const Vector* b,
    const bool* corr,
    Matrix3x3* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        if(corr[id])
        {
            C[id](0,0) = a[id].x * b[id].x;
            C[id](1,0) = a[id].x * b[id].y;
            C[id](2,0) = a[id].x * b[id].z;
            C[id](0,1) = a[id].y * b[id].x;
            C[id](1,1) = a[id].y * b[id].y;
            C[id](2,1) = a[id].y * b[id].z;
            C[id](0,2) = a[id].z * b[id].x;
            C[id](1,2) = a[id].z * b[id].y;
            C[id](2,2) = a[id].z * b[id].z;
        } else {
            C[id].setZeros();
        }
    }
}

template<unsigned int blockSize, typename T>
__device__ void warpReduce(volatile T* sdata, unsigned int tid)
{
    if(blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if(blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if(blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if(blockSize >=  8) sdata[tid] += sdata[tid + 4];
    if(blockSize >=  4) sdata[tid] += sdata[tid + 2];
    if(blockSize >=  2) sdata[tid] += sdata[tid + 1];
}


template<unsigned int blockSize, typename T>
__global__ void sum_kernel(
    const T* data,
    T* res,
    unsigned int N)
{
    __shared__ T sdata[blockSize];
    
    const unsigned int tid = threadIdx.x;
    const unsigned int globId = N * blockIdx.x + threadIdx.x;
    const unsigned int rows = (N + blockSize - 1) / blockSize;

    sdata[tid] *= 0.0;
    for(unsigned int i=0; i<rows; i++)
    {
        if(tid + blockSize * i < N)
        {
            sdata[threadIdx.x] += data[globId + blockSize * i];
        }
    }
    __syncthreads();

    for(unsigned int s = blockSize / 2; s > 32; s >>= 1)
    {
        if(tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if(tid < blockSize / 2 && tid < 32)
    {
        warpReduce<blockSize>(sdata, tid);
    }

    if(tid == 0)
    {
        res[blockIdx.x] = sdata[0];
    }
}

template<unsigned int blockSize>
__global__ void cov_kernel(
    const Vector* v1,
    const Vector* v2,
    Matrix3x3* res,
    unsigned int N)
{
    __shared__ Matrix3x3 sdata[blockSize];
    
    const unsigned int tid = threadIdx.x;
    const unsigned int globId = N * blockIdx.x + threadIdx.x;
    const unsigned int rows = (N + blockSize - 1) / blockSize;

    sdata[tid].setZeros();
    for(unsigned int i=0; i<rows; i++)
    {
        if(tid + blockSize * i < N)
        {
            const Vector& a = v1[globId + blockSize * i];
            const Vector& b = v2[globId + blockSize * i];
            sdata[tid](0,0) += a.x * b.x;
            sdata[tid](1,0) += a.x * b.y;
            sdata[tid](2,0) += a.x * b.z;
            sdata[tid](0,1) += a.y * b.x;
            sdata[tid](1,1) += a.y * b.y;
            sdata[tid](2,1) += a.y * b.z;
            sdata[tid](0,2) += a.z * b.x;
            sdata[tid](1,2) += a.z * b.y;
            sdata[tid](2,2) += a.z * b.z;
        }
    }
    __syncthreads();

    for(unsigned int s = blockSize / 2; s > 32; s >>= 1)
    {
        if(tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if(tid < blockSize / 2 && tid < 32)
    {
        warpReduce<blockSize>(sdata, tid);
    }

    if(tid == 0)
    {
        res[blockIdx.x] = sdata[0] / static_cast<float>(N);
    }
}

////////////
// #multNxN
void multNxN(
    const Memory<Quaternion, VRAM_CUDA>& A,
    const Memory<Quaternion, VRAM_CUDA>& B,
    Memory<Quaternion, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    multNxN_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
}

Memory<Quaternion, VRAM_CUDA> multNxN(
    const Memory<Quaternion, VRAM_CUDA>& A, 
    const Memory<Quaternion, VRAM_CUDA>& B)
{
    Memory<Quaternion, VRAM_CUDA> C(A.size());
    // mult
    multNxN(A, B, C);
    return C;
}

void multNxN(
    const Memory<Quaternion, VRAM_CUDA>& A,
    const Memory<Vector, VRAM_CUDA>& b, 
    Memory<Vector, VRAM_CUDA>& c)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    multNxN_kernel<<<gridSize, blockSize>>>(A.raw(), b.raw(), c.raw(), A.size());
}

Memory<Vector, VRAM_CUDA> multNxN(
    const Memory<Quaternion, VRAM_CUDA>& A,
    const Memory<Vector, VRAM_CUDA>& b)
{
    Memory<Vector, VRAM_CUDA> c(A.size());
    multNxN(A, b, c);
    return c;
}

/// TRANSFORM
void multNxN(
    const Memory<Transform, VRAM_CUDA>& T1,
    const Memory<Transform, VRAM_CUDA>& T2,
    Memory<Transform, VRAM_CUDA>& Tr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (T1.size() + blockSize - 1) / blockSize;
    multNxN_kernel<<<gridSize, blockSize>>>(T1.raw(), T2.raw(), Tr.raw(), T1.size());
}

Memory<Transform, VRAM_CUDA> multNxN(
    const Memory<Transform, VRAM_CUDA>& T1,
    const Memory<Transform, VRAM_CUDA>& T2)
{
    Memory<Transform, VRAM_CUDA> Tr(T1.size());
    multNxN(T1,T2,Tr);
    return Tr;
}

void multNxN(
    const Memory<Transform, VRAM_CUDA>& T,
    const Memory<Vector, VRAM_CUDA>& x,
    Memory<Vector, VRAM_CUDA>& c)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (T.size() + blockSize - 1) / blockSize;
    multNxN_kernel<<<gridSize, blockSize>>>(T.raw(), x.raw(), c.raw(), T.size());
}

Memory<Vector, VRAM_CUDA> multNxN(
    const Memory<Transform, VRAM_CUDA>& T,
    const Memory<Vector, VRAM_CUDA>& x)
{
    Memory<Vector, VRAM_CUDA> c(T.size());
    multNxN(T,x,c);
    return c;
}

void multNxN(
    const Memory<Matrix3x3, VRAM_CUDA>& M1,
    const Memory<Matrix3x3, VRAM_CUDA>& M2,
    Memory<Matrix3x3, VRAM_CUDA>& Mr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M1.size() + blockSize - 1) / blockSize;
    multNxN_kernel<<<gridSize, blockSize>>>(M1.raw(), M2.raw(), Mr.raw(), M1.size());
}

Memory<Matrix3x3, VRAM_CUDA> multNxN(
    const Memory<Matrix3x3, VRAM_CUDA>& M1,
    const Memory<Matrix3x3, VRAM_CUDA>& M2)
{
    Memory<Matrix3x3, VRAM_CUDA> Mr(M1.size());
    multNxN(M1,M2,Mr);
    return Mr;
}

void multNxN(
    const Memory<Matrix3x3, VRAM_CUDA>& M,
    const Memory<Vector, VRAM_CUDA>& x,
    Memory<Vector, VRAM_CUDA>& c)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M.size() + blockSize - 1) / blockSize;
    multNxN_kernel<<<gridSize, blockSize>>>(M.raw(), x.raw(), c.raw(), M.size());
}

Memory<Vector, VRAM_CUDA> multNxN(
    const Memory<Matrix3x3, VRAM_CUDA>& M,
    const Memory<Vector, VRAM_CUDA>& x)
{
    Memory<Vector, VRAM_CUDA> c(M.size());
    multNxN(M, x, c);
    return c;
}

////////
// #multNx1
///
void multNx1(
    const Memory<Quaternion, VRAM_CUDA>& A,
    const Memory<Quaternion, VRAM_CUDA>& b,
    Memory<Quaternion, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    multNxN_kernel<<<gridSize, blockSize>>>(A.raw(), b.raw(), C.raw(), A.size());
}

Memory<Quaternion, VRAM_CUDA> multNx1(
    const Memory<Quaternion, VRAM_CUDA>& A, 
    const Memory<Quaternion, VRAM_CUDA>& b)
{
    Memory<Quaternion, VRAM_CUDA> C(A.size());
    // mult
    multNx1(A, b, C);
    return C;
}

void multNx1(
    const Memory<Quaternion, VRAM_CUDA>& A,
    const Memory<Vector, VRAM_CUDA>& b, 
    Memory<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    multNx1_kernel<<<gridSize, blockSize>>>(A.raw(), b.raw(), C.raw(), A.size());
}

Memory<Vector, VRAM_CUDA> multNx1(
    const Memory<Quaternion, VRAM_CUDA>& A,
    const Memory<Vector, VRAM_CUDA>& b)
{
    Memory<Vector, VRAM_CUDA> C(A.size());
    multNx1(A, b, C);
    return C;
}

void multNx1(
    const Memory<Transform, VRAM_CUDA>& T1,
    const Memory<Transform, VRAM_CUDA>& t2,
    Memory<Transform, VRAM_CUDA>& Tr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (T1.size() + blockSize - 1) / blockSize;
    multNx1_kernel<<<gridSize, blockSize>>>(T1.raw(), t2.raw(), Tr.raw(), T1.size());
}

Memory<Transform, VRAM_CUDA> multNx1(
    const Memory<Transform, VRAM_CUDA>& T1,
    const Memory<Transform, VRAM_CUDA>& t2)
{
    Memory<Transform, VRAM_CUDA> Tr(T1.size());
    multNx1(T1,t2,Tr);
    return Tr;
}

void multNx1(
    const Memory<Transform, VRAM_CUDA>& T,
    const Memory<Vector, VRAM_CUDA>& x,
    Memory<Vector, VRAM_CUDA>& c)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (T.size() + blockSize - 1) / blockSize;
    multNx1_kernel<<<gridSize, blockSize>>>(T.raw(), x.raw(), c.raw(), T.size());
}

Memory<Vector, VRAM_CUDA> multNx1(
    const Memory<Transform, VRAM_CUDA>& T,
    const Memory<Vector, VRAM_CUDA>& x)
{
    Memory<Vector, VRAM_CUDA> C(T.size());
    multNx1(T,x,C);
    return C;
}

void multNx1(
    const Memory<Matrix3x3, VRAM_CUDA>& M1,
    const Memory<Matrix3x3, VRAM_CUDA>& m2,
    Memory<Matrix3x3, VRAM_CUDA>& Mr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M1.size() + blockSize - 1) / blockSize;
    multNx1_kernel<<<gridSize, blockSize>>>(M1.raw(), m2.raw(), Mr.raw(), M1.size());
}

Memory<Matrix3x3, VRAM_CUDA> multNx1(
    const Memory<Matrix3x3, VRAM_CUDA>& M1,
    const Memory<Matrix3x3, VRAM_CUDA>& m2)
{
    Memory<Matrix3x3, VRAM_CUDA> Mr(M1.size());
    multNx1(M1,m2,Mr);
    return Mr;
}

void multNx1(
    const Memory<Matrix3x3, VRAM_CUDA>& M,
    const Memory<Vector, VRAM_CUDA>& x,
    Memory<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M.size() + blockSize - 1) / blockSize;
    multNx1_kernel<<<gridSize, blockSize>>>(M.raw(), x.raw(), C.raw(), M.size());
}

Memory<Vector, VRAM_CUDA> multNx1(
    const Memory<Matrix3x3, VRAM_CUDA>& M,
    const Memory<Vector, VRAM_CUDA>& x)
{
    Memory<Vector, VRAM_CUDA> c(M.size());
    multNx1(M, x, c);
    return c;
}

/////////////
// #mult1xN
////////
void mult1xN(
    const Memory<Quaternion, VRAM_CUDA>& a,
    const Memory<Quaternion, VRAM_CUDA>& B,
    Memory<Quaternion, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (B.size() + blockSize - 1) / blockSize;
    mult1xN_kernel<<<gridSize, blockSize>>>(a.raw(), B.raw(), C.raw(), B.size());
}

Memory<Quaternion, VRAM_CUDA> mult1xN(
    const Memory<Quaternion, VRAM_CUDA>& a, 
    const Memory<Quaternion, VRAM_CUDA>& B)
{
    Memory<Quaternion, VRAM_CUDA> C(B.size());
    mult1xN(a, B, C);
    return C;
}

void mult1xN(
    const Memory<Quaternion, VRAM_CUDA>& a,
    const Memory<Vector, VRAM_CUDA>& B, 
    Memory<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (B.size() + blockSize - 1) / blockSize;
    mult1xN_kernel<<<gridSize, blockSize>>>(a.raw(), B.raw(), C.raw(), B.size());
}

Memory<Vector, VRAM_CUDA> mult1xN(
    const Memory<Quaternion, VRAM_CUDA>& a,
    const Memory<Vector, VRAM_CUDA>& B)
{
    Memory<Vector, VRAM_CUDA> C(B.size());
    mult1xN(a, B, C);
    return C;
}

void mult1xN(
    const Memory<Transform, VRAM_CUDA>& t1,
    const Memory<Transform, VRAM_CUDA>& T2,
    Memory<Transform, VRAM_CUDA>& Tr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (T2.size() + blockSize - 1) / blockSize;
    mult1xN_kernel<<<gridSize, blockSize>>>(t1.raw(), T2.raw(), Tr.raw(), T2.size());
}

Memory<Transform, VRAM_CUDA> mult1xN(
    const Memory<Transform, VRAM_CUDA>& t1,
    const Memory<Transform, VRAM_CUDA>& T2)
{
    Memory<Transform, VRAM_CUDA> Tr(T2.size());
    mult1xN(t1, T2, Tr);
    return Tr;
}

void mult1xN(
    const Memory<Transform, VRAM_CUDA>& t,
    const Memory<Vector, VRAM_CUDA>& X,
    Memory<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (X.size() + blockSize - 1) / blockSize;
    mult1xN_kernel<<<gridSize, blockSize>>>(t.raw(), X.raw(), C.raw(), X.size());
}

Memory<Vector, VRAM_CUDA> mult1xN(
    const Memory<Transform, VRAM_CUDA>& t,
    const Memory<Vector, VRAM_CUDA>& X)
{
    Memory<Vector, VRAM_CUDA> C(X.size());
    mult1xN(t, X, C);
    return C;
}

void mult1xN(
    const Memory<Matrix3x3, VRAM_CUDA>& m1,
    const Memory<Matrix3x3, VRAM_CUDA>& M2,
    Memory<Matrix3x3, VRAM_CUDA>& Mr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M2.size() + blockSize - 1) / blockSize;
    mult1xN_kernel<<<gridSize, blockSize>>>(m1.raw(), M2.raw(), Mr.raw(), M2.size());
}

Memory<Matrix3x3, VRAM_CUDA> mult1xN(
    const Memory<Matrix3x3, VRAM_CUDA>& m1,
    const Memory<Matrix3x3, VRAM_CUDA>& M2)
{
    Memory<Matrix3x3, VRAM_CUDA> Mr(M2.size());
    mult1xN(m1, M2, Mr);
    return Mr;
}

void mult1xN(
    const Memory<Matrix3x3, VRAM_CUDA>& m,
    const Memory<Vector, VRAM_CUDA>& X,
    Memory<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (X.size() + blockSize - 1) / blockSize;
    mult1xN_kernel<<<gridSize, blockSize>>>(m.raw(), X.raw(), C.raw(), X.size());
}

Memory<Vector, VRAM_CUDA> mult1xN(
    const Memory<Matrix3x3, VRAM_CUDA>& m,
    const Memory<Vector, VRAM_CUDA>& X)
{
    Memory<Vector, VRAM_CUDA> C(X.size());
    mult1xN(m, X, C);
    return C;
}

///////
// #add
void addNxN(
    const Memory<Vector, VRAM_CUDA>& A,
    const Memory<Vector, VRAM_CUDA>& B,
    Memory<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    addNxN_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
}

Memory<Vector, VRAM_CUDA> addNxN(
    const Memory<Vector, VRAM_CUDA>& A,
    const Memory<Vector, VRAM_CUDA>& B)
{
    Memory<Vector, VRAM_CUDA> C(A.size());
    addNxN(A, B, C);
    return C;
}

////////
// #sub
void subNxN(
    const Memory<Vector, VRAM_CUDA>& A,
    const Memory<Vector, VRAM_CUDA>& B,
    Memory<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    subNxN_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
}

Memory<Vector, VRAM_CUDA> subNxN(
    const Memory<Vector, VRAM_CUDA>& A,
    const Memory<Vector, VRAM_CUDA>& B)
{
    Memory<Vector, VRAM_CUDA> C(A.size());
    subNxN(A, B, C);
    return C;
}

void subNx1(
    const Memory<Vector, VRAM_CUDA>& A,
    const Memory<Vector, VRAM_CUDA>& b,
    Memory<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    subNx1_kernel<<<gridSize, blockSize>>>(A.raw(), b.raw(), C.raw(), A.size());
}

Memory<Vector, VRAM_CUDA> subNx1(
    const Memory<Vector, VRAM_CUDA>& A,
    const Memory<Vector, VRAM_CUDA>& b)
{
    Memory<Vector, VRAM_CUDA> C(A.size());
    subNx1(A, b, C);
    return C;
}

/////
// #transpose
void transpose(
    const Memory<Matrix3x3, VRAM_CUDA>& A, 
    Memory<Matrix3x3, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    transpose_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
}

Memory<Matrix3x3, VRAM_CUDA> transpose(
    const Memory<Matrix3x3, VRAM_CUDA>& A)
{
    Memory<Matrix3x3, VRAM_CUDA> B(A.size());
    transpose(A, B);
    return B;
}

void transpose(
    const Memory<Matrix4x4, VRAM_CUDA>& A,
    Memory<Matrix4x4, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    transpose_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
}

Memory<Matrix4x4, VRAM_CUDA> transpose(
    const Memory<Matrix4x4, VRAM_CUDA>& A)
{
    Memory<Matrix4x4, VRAM_CUDA> B(A.size());
    transpose(A, B);
    return B;
}

///////
// #transposeInplace
void transposeInplace(
    Memory<Matrix3x3, VRAM_CUDA>& A)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    transposeInplace_kernel<<<gridSize, blockSize>>>(A.raw(), A.size());
}

//////
// #invert
void invert(
    const Memory<Matrix3x3, VRAM_CUDA>& A, 
    Memory<Matrix3x3, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    invert_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
}

Memory<Matrix3x3, VRAM_CUDA> invert(
    const Memory<Matrix3x3, VRAM_CUDA>& A)
{
    Memory<Matrix3x3, VRAM_CUDA> B(A.size());
    invert(A, B);
    return B;
}

void invert(
    const Memory<Matrix4x4, VRAM_CUDA>& A,
    Memory<Matrix4x4, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    invert_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
}

Memory<Matrix4x4, VRAM_CUDA> invert(
    const Memory<Matrix4x4, VRAM_CUDA>& A)
{
    Memory<Matrix4x4, VRAM_CUDA> B(A.size());
    invert(A, B);
    return B;
}

void invert(
    const Memory<Transform, VRAM_CUDA>& A,
    Memory<Transform, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    invert_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
}

Memory<Transform, VRAM_CUDA> invert(
    const Memory<Transform, VRAM_CUDA>& A)
{
    Memory<Transform, VRAM_CUDA> B(A.size());
    invert(A, B);
    return B;
}

//////
// #divNxN
void divNxN(
    const Memory<Vector, VRAM_CUDA>& A, 
    const Memory<unsigned int, VRAM_CUDA>& B,
    Memory<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    divNxN_conv_kernel<float><<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
}

Memory<Vector, VRAM_CUDA> divNxN(
    const Memory<Vector, VRAM_CUDA>& A, 
    const Memory<unsigned int, VRAM_CUDA>& B)
{
    Memory<Vector, VRAM_CUDA> C(A.size());
    divNxN(A, B, C);
    return C;
}

void divNxN(
    const Memory<Matrix3x3, VRAM_CUDA>& A, 
    const Memory<unsigned int, VRAM_CUDA>& B, 
    Memory<Matrix3x3, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    divNxN_conv_kernel<float><<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
}

Memory<Matrix3x3, VRAM_CUDA> divNxN(
    const Memory<Matrix3x3, VRAM_CUDA>& A,
    const Memory<unsigned int, VRAM_CUDA>& B)
{
    Memory<Matrix3x3, VRAM_CUDA> C(A.size());
    divNxN(A, B, C);
    return C;
}

///////
// #divNxNIgnoreZeros
void divNxNIgnoreZeros(
    const Memory<Vector, VRAM_CUDA>& A, 
    const Memory<unsigned int, VRAM_CUDA>& B,
    Memory<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    divNxNIgnoreZeros_conv_kernel<float><<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
}

Memory<Vector, VRAM_CUDA> divNxNIgnoreZeros(
    const Memory<Vector, VRAM_CUDA>& A, 
    const Memory<unsigned int, VRAM_CUDA>& B)
{
    Memory<Vector, VRAM_CUDA> C(A.size());
    divNxNIgnoreZeros(A, B, C);
    return C;
}

void divNxNIgnoreZeros(
    const Memory<Matrix3x3, VRAM_CUDA>& A, 
    const Memory<unsigned int, VRAM_CUDA>& B,
    Memory<Matrix3x3, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    divNxNIgnoreZeros_conv_kernel<float><<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
}

Memory<Matrix3x3, VRAM_CUDA> divNxNIgnoreZeros(
    const Memory<Matrix3x3, VRAM_CUDA>& A, 
    const Memory<unsigned int, VRAM_CUDA>& B)
{
    Memory<Matrix3x3, VRAM_CUDA> C(A.size());
    divNxNIgnoreZeros(A, B, C);
    return C;
}

void divNxNIgnoreZeros(
    const Memory<float, VRAM_CUDA>& A, 
    const Memory<unsigned int, VRAM_CUDA>& B,
    Memory<float, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    divNxNIgnoreZeros_conv_kernel<float><<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
}

Memory<float, VRAM_CUDA> divNxNIgnoreZeros(
    const Memory<float, VRAM_CUDA>& A, 
    const Memory<unsigned int, VRAM_CUDA>& B)
{
    Memory<float, VRAM_CUDA> C(A.size());
    divNxNIgnoreZeros(A, B, C);
    return C;
}

////////
// #divNxNInplace
void divNxNInplace(
    Memory<Vector, VRAM_CUDA>& A, 
    const Memory<float, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    divNxNInplace_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
}

void divNxNInplace(
    Memory<Matrix3x3, VRAM_CUDA>& A, 
    const Memory<unsigned int, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    divNxNInplace_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
}

////////
// #divNx1Inplace
void divNx1Inplace(
    Memory<Matrix3x3, VRAM_CUDA>& A, 
    const unsigned int& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    divNx1Inplace_kernel<<<gridSize, blockSize>>>(A.raw(), B, A.size());
}

void divNx1Inplace(
    Memory<Vector, VRAM_CUDA>& A, 
    const unsigned int& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    divNx1Inplace_kernel<<<gridSize, blockSize>>>(A.raw(), B, A.size());
}


////////
// #convert
void convert(
    const Memory<uint8_t, VRAM_CUDA>& from, 
    Memory<float, VRAM_CUDA>& to)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (from.size() + blockSize - 1) / blockSize;
    convert_kernel<<<gridSize, blockSize>>>(from.raw(), to.raw(), from.size());
}

void convert(
    const Memory<bool, VRAM_CUDA>& from, 
    Memory<unsigned int, VRAM_CUDA>& to)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (from.size() + blockSize - 1) / blockSize;
    convert_kernel<<<gridSize, blockSize>>>(from.raw(), to.raw(), from.size());
}

void copy(const Memory<unsigned int, VRAM_CUDA>& from, 
    Memory<bool, VRAM_CUDA>& to)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (from.size() + blockSize - 1) / blockSize;
    convert_kernel<<<gridSize, blockSize>>>(from.raw(), to.raw(), from.size());
}

////////
// #pack
void pack(
    const Memory<Matrix3x3, VRAM_CUDA>& R,
    const Memory<Vector, VRAM_CUDA>& t,
    Memory<Transform, VRAM_CUDA>& T)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (R.size() + blockSize - 1) / blockSize;
    pack_kernel<<<gridSize, blockSize>>>(R.raw(), t.raw(), T.raw(), R.size());
}

void pack(
    const Memory<Quaternion, VRAM_CUDA>& R,
    const Memory<Vector, VRAM_CUDA>& t,
    Memory<Transform, VRAM_CUDA>& T)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (R.size() + blockSize - 1) / blockSize;
    pack_kernel<<<gridSize, blockSize>>>(R.raw(), t.raw(), T.raw(), R.size());
}

////////
// #multNxNTransposed
void multNxNTransposed(
    const Memory<Vector, VRAM_CUDA>& m1,
    const Memory<Vector, VRAM_CUDA>& m2,
    Memory<Matrix3x3, VRAM_CUDA>& Cs)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (m1.size() + blockSize - 1) / blockSize;
    covParts_kernel<<<gridSize, blockSize>>>(m1.raw(), m2.raw(), Cs.raw(), m1.size());
}

Memory<Matrix3x3, VRAM_CUDA> multNxNTransposed(
    const Memory<Vector, VRAM_CUDA>& m1,
    const Memory<Vector, VRAM_CUDA>& m2)
{
    Memory<Matrix3x3, VRAM_CUDA> Cs(m1.size());
    multNxNTransposed(m1, m2, Cs);
    return Cs;
}

void multNxNTransposed(
    const Memory<Vector, VRAM_CUDA>& m1,
    const Memory<Vector, VRAM_CUDA>& m2,
    const Memory<bool, VRAM_CUDA>& mask,
    Memory<Matrix3x3, VRAM_CUDA>& Cs)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (m1.size() + blockSize - 1) / blockSize;
    covParts_kernel<<<gridSize, blockSize>>>(m1.raw(), m2.raw(), mask.raw(), Cs.raw(), m1.size());
}
    
Memory<Matrix3x3, VRAM_CUDA> multNxNTransposed(
    const Memory<Vector, VRAM_CUDA>& m1,
    const Memory<Vector, VRAM_CUDA>& m2,
    const Memory<bool, VRAM_CUDA>& mask)
{
    Memory<Matrix3x3, VRAM_CUDA> Cs(m1.size());
    multNxNTransposed(m1, m2, mask, Cs);
    return Cs;
}

//////////
// #sum
// TODO: check perfomance of sum_kernel
void sum(
    const Memory<Vector, VRAM_CUDA>& data,
    Memory<Vector, VRAM_CUDA>& s)
{
    sum_kernel<1024> <<<1, 1024>>>(data.raw(), s.raw(), data.size() );
}

Memory<Vector, VRAM_CUDA> sum(
    const Memory<Vector, VRAM_CUDA>& data)
{
    Memory<Vector, VRAM_CUDA> s(1);
    sum(data, s);
    return s;
}

//////////
// #mean
void mean(
    const Memory<Vector, VRAM_CUDA>& X,
    Memory<Vector, VRAM_CUDA>& res)
{
    sum(X, res);
    divNx1Inplace(res, X.size());
}

Memory<Vector, VRAM_CUDA> mean(
    const Memory<Vector, VRAM_CUDA>& X)
{
    Memory<Vector, VRAM_CUDA> res(1);
    mean(X, res);
    return res;
}

//////////
// #cov
void cov(
    const Memory<Vector, VRAM_CUDA>& v1,
    const Memory<Vector, VRAM_CUDA>& v2,
    Memory<Matrix3x3, VRAM_CUDA>& C)
{
    cov_kernel<1024> <<<1, 1024>>>(v1.raw(), v2.raw(), C.raw(), v1.size() );
}

Memory<Matrix3x3, VRAM_CUDA> cov(
    const Memory<Vector, VRAM_CUDA>& v1,
    const Memory<Vector, VRAM_CUDA>& v2
)
{
    Memory<Matrix3x3, VRAM_CUDA> C(1);
    cov(v1, v2, C);
    return C;
}


} // namespace rmagine