#include "hip/hip_runtime.h"
#include <optix.h>
#include <hip/hip_math_constants.h>

#include "rmagine/math/math.h"
#include "rmagine/simulation/optix/OptixSimulationData.hpp"

using namespace rmagine;

extern "C" {
__constant__ OptixSimulationDataNormalsOnDn mem;
}

extern "C" __global__ void __raygen__rg()
{
    // Lookup our location within the launch grid
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    // vertical id
    const unsigned int hid = idx.x;
    // horizontal id
    const unsigned int vid = idx.y;
    // pose id
    const unsigned int pid = idx.z;

    const unsigned int loc_id = mem.model->getBufferId(vid, hid);
    const unsigned int glob_id = pid * mem.model->size() + loc_id;
    
    const Transform Tsm = mem.Tbm[pid] * mem.Tsb[0];

    const Vector ray_orig_s = mem.model->getOrigin(vid, hid);
    const Vector ray_dir_s = mem.model->getDirection(vid, hid);

    const Vector ray_orig_m = Tsm * ray_orig_s;
    const Vector ray_dir_m = Tsm.R * ray_dir_s;
    

    unsigned int p0, p1, p2, p3, p4, p5, p6, p7;
    
    p0 = glob_id;
    p1 = __float_as_uint(Tsm.R.x);
    p2 = __float_as_uint(Tsm.R.y);
    p3 = __float_as_uint(Tsm.R.z);
    p4 = __float_as_uint(Tsm.R.w);
    p5 = __float_as_uint(Tsm.t.x);
    p6 = __float_as_uint(Tsm.t.y);
    p7 = __float_as_uint(Tsm.t.z);

    optixTrace(
            mem.handle,
            make_float3(ray_orig_m.x, ray_orig_m.y, ray_orig_m.z ),
            make_float3(ray_dir_m.x, ray_dir_m.y, ray_dir_m.z),
            0.0f,               // Min intersection distance
            mem.model->range.max,                   // Max intersection distance
            0.0f,                       // rayTime -- used for motion blur
            OptixVisibilityMask( 1 ),   // Specify always visible
            OPTIX_RAY_FLAG_DISABLE_ANYHIT,
            0,          // SBT offset
            1,          // SBT stride
            0,          // missSBTIndex
            p0, p1, p2, p3, p4, p5, p6, p7 );
}

extern "C" __global__ void __miss__ms()
{
    const unsigned int glob_id = optixGetPayload_0();
    mem.normals[glob_id] = {
        mem.model->range.max + 1.0f,
        mem.model->range.max + 1.0f,
        mem.model->range.max + 1.0f
    };
}

extern "C" __global__ void __closesthit__ch()
{
    // Get Payloads
    const unsigned int glob_id = optixGetPayload_0();
    Transform Tsm;
    Tsm.R.x = __uint_as_float(optixGetPayload_1());
    Tsm.R.y = __uint_as_float(optixGetPayload_2());
    Tsm.R.z = __uint_as_float(optixGetPayload_3());
    Tsm.R.w = __uint_as_float(optixGetPayload_4());
    Tsm.t.x = __uint_as_float(optixGetPayload_5());
    Tsm.t.y = __uint_as_float(optixGetPayload_6());
    Tsm.t.z = __uint_as_float(optixGetPayload_7());
    const Transform Tms = Tsm.inv();

    // Get additional info
    const unsigned int face_id = optixGetPrimitiveIndex();
    const unsigned int object_id = optixGetInstanceIndex();
    
    const float3 dir_m = optixGetWorldRayDirection();
    const Vector ray_dir_m{dir_m.x, dir_m.y, dir_m.z};
    const Vector ray_dir_s = Tms.R * ray_dir_m;

    rmagine::HitGroupDataScene* hg_data  = reinterpret_cast<rmagine::HitGroupDataScene*>( optixGetSbtDataPointer() );
    
    const int mesh_id = hg_data->inst_to_mesh[object_id];
    const MeshAttributes* mesh_attr = &hg_data->mesh_attributes[mesh_id];

    const float3 normal = make_float3(
        mesh_attr->face_normals[face_id].x, 
        mesh_attr->face_normals[face_id].y, 
        mesh_attr->face_normals[face_id].z);
    const float3 normal_world = optixTransformNormalFromObjectToWorldSpace(normal);

    Vector nint{normal_world.x, normal_world.y, normal_world.z};
    nint.normalize();
    nint = Tms.R * nint;

    // flip?
    if(ray_dir_s.dot(nint) > 0.0)
    {
        nint *= -1.0;
    }

    mem.normals[glob_id] = nint.normalized();
}