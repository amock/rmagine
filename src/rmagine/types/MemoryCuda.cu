#include "rmagine/types/MemoryCuda.hpp"

#include <hip/hip_runtime.h>

namespace rmagine {

// CUDA HELPER
namespace cuda {

void* memcpyHostToDevice(void* dest, const void* src, std::size_t count)
{
    CUDA_DEBUG( hipMemcpy(dest, src, count, hipMemcpyHostToDevice) );
    return dest;
}

void* memcpyDeviceToHost(void* dest, const void* src, std::size_t count)
{
    CUDA_DEBUG( hipMemcpy(dest, src, count, hipMemcpyDeviceToHost) );
    return dest;
}

void* memcpyDeviceToDevice(void* dest, const void* src, std::size_t count)
{
    CUDA_DEBUG( hipMemcpy(dest, src, count, hipMemcpyDeviceToDevice) );
    return dest;
}

void* memcpyHostToHost(void* dest, const void* src, std::size_t count)
{
    CUDA_DEBUG( hipMemcpy(dest, src, count, hipMemcpyHostToHost) );
    return dest;
}

} // namespace cuda

} // namespace mamcl