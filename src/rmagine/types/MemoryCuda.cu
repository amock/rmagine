#include "rmagine/types/MemoryCuda.hpp"
#include "rmagine/util/cuda/CudaStream.hpp"

#include <hip/hip_runtime.h>


namespace rmagine {

// CUDA HELPER
namespace cuda {

void* memcpyHostToDevice(void* dest, const void* src, std::size_t count)
{
    RM_CUDA_CHECK( hipMemcpy(dest, src, count, hipMemcpyHostToDevice) );
    return dest;
}

void* memcpyHostToDevice(void* dest, const void* src, std::size_t count, CudaStreamPtr stream)
{
    RM_CUDA_CHECK( hipMemcpyAsync(dest, src, count, hipMemcpyHostToDevice, stream->handle()) );
    return dest;
}

void* memcpyDeviceToHost(void* dest, const void* src, std::size_t count)
{
    RM_CUDA_CHECK( hipMemcpy(dest, src, count, hipMemcpyDeviceToHost) );
    return dest;
}

void* memcpyDeviceToHost(void* dest, const void* src, std::size_t count, CudaStreamPtr stream)
{
    RM_CUDA_CHECK( hipMemcpyAsync(dest, src, count, hipMemcpyDeviceToHost, stream->handle()) );
    return dest;
}

void* memcpyDeviceToDevice(void* dest, const void* src, std::size_t count)
{
    RM_CUDA_CHECK( hipMemcpy(dest, src, count, hipMemcpyDeviceToDevice) );
    return dest;
}

void* memcpyDeviceToDevice( void* dest, const void* src, std::size_t count, CudaStreamPtr stream)
{
    RM_CUDA_CHECK( hipMemcpyAsync(dest, src, count, hipMemcpyDeviceToDevice, stream->handle()) );
    return dest;
}

void* memcpyHostToHost(void* dest, const void* src, std::size_t count)
{
    RM_CUDA_CHECK( hipMemcpy(dest, src, count, hipMemcpyHostToHost) );
    return dest;
}

void* memcpyHostToHost(     void* dest, const void* src, std::size_t count, CudaStreamPtr stream)
{
    RM_CUDA_CHECK( hipMemcpyAsync(dest, src, count, hipMemcpyHostToHost, stream->handle()) );
    return dest;
}

} // namespace cuda

} // namespace mamcl