#include "hip/hip_runtime.h"
#include "rmagine/math/memory_math.cuh"
#include "rmagine/math/types.h"
#include "rmagine/math/linalg.cuh"
#include "rmagine/util/cuda/CudaDebug.hpp"

namespace rmagine 
{

namespace cuda
{

////////
// Generic Kernel
///
template<typename In1T, typename In2T, typename ResT>
__global__ void multNxN_kernel(
    const In1T* A,
    const In2T* B,
    ResT* C,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] * B[id];
    }
}

template<typename In1T, typename In2T, typename ResT>
__global__ void multNxN_conv_kernel(
    const In1T* A,
    const In2T* B,
    ResT* C,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id].set(A[id] * B[id]);
    }
}

template<typename In1T, typename In2T, typename ResT>
__global__ void multNx1_kernel(
    const In1T* A,
    const In2T* b,
    ResT* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] * b[0];
    }
}

template<typename In1T, typename In2T, typename ResT>
__global__ void mult1xN_kernel(
    const In1T* a,
    const In2T* B,
    ResT* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = a[0] * B[id];
    }
}


template<typename In1T, typename In2T, typename ResT>
__global__ void addNxN_kernel(
    const In1T* A,
    const In2T* B,
    ResT* C,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] + B[id];
    }
}

template<typename In1T, typename In2T, typename ResT>
__global__ void subNxN_kernel(
    const In1T* A,
    const In2T* B,
    ResT* C,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] - B[id];
    }
}

template<typename In1T, typename In2T, typename ResT>
__global__ void subNx1_kernel(
    const In1T* A,
    const In2T* b,
    ResT* C,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] - b[0];
    }
}


template<typename T>
__global__ void transpose_kernel(
    const T* A,
    T* B,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        B[id] = A[id].transpose();
    }
}

template<typename T>
__global__ void transposeInplace_kernel(
    T* A,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        A[id].transposeInplace();
    }
}


template<typename T>
__global__ void invert_kernel(
    const T* A,
    T* B,
    unsigned int N
)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        B[id] = A[id].inv();
    }
}

template<typename In1T, typename In2T, typename ResT>
__global__ void divNxN_kernel(
    const In1T* A, 
    const In2T* B,
    ResT* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] / B[id];
    }
}

template<typename ConvT, typename In1T, typename In2T, typename ResT>
__global__ void divNxN_conv_kernel(
    const In1T* A, 
    const In2T* B,
    ResT* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id] = A[id] / static_cast<ConvT>(B[id]);
    }
}

template<typename ConvT>
__global__ void divNxNIgnoreZeros_conv_kernel(
    const Vector* A, 
    const unsigned int* B,
    Vector* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        if(B[id] > 0)
        {
            C[id] = A[id] / static_cast<ConvT>(B[id]);
        } else {
            C[id].setZeros();
        }
    }
}

template<typename ConvT>
__global__ void divNxNIgnoreZeros_conv_kernel(
    const Matrix3x3* A,
    const unsigned int* B,
    Matrix3x3* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        if(B[id] > 0)
        {
            C[id] = A[id] / static_cast<ConvT>(B[id]);
        } else {
            C[id].setZeros();
        }
    }
}

template<typename ConvT>
__global__ void divNxNIgnoreZeros_conv_kernel(
    const float* A, 
    const unsigned int* B,
    float* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        if(B[id] > 0)
        {
            C[id] = A[id] / static_cast<ConvT>(B[id]);
        } else {
            C[id] = 0.0;
        }
    }
}

__global__ void divNxNInplace_kernel(
    Vector* A, 
    const float* B,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        A[id] /= B[id];
    }
}

__global__ void divNxNInplace_kernel(
    Matrix3x3* A, 
    const unsigned int* B,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        A[id] /= static_cast<float>(B[id]);
    }
}

template<typename T>
__global__ void divNx1Inplace_kernel(
    T* A,
    unsigned int b,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        A[id] /= static_cast<float>(b);
    }
}

__global__ void convert_kernel(const uint8_t* from, float* to, unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        to[id] = static_cast<float>(from[id]);
    }
}

__global__
void convert_kernel(const bool* from, unsigned int* to, unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        to[id] = static_cast<unsigned int>(from[id]);
    }
}

__global__
void convert_kernel(const unsigned int* from, bool* to, unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        to[id] = (from[id] > 0);
    }
}

__global__ void pack_kernel(
    const Matrix3x3* R, 
    const Vector* t, // Vector3d / Vector3f
    Transform* T,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        T[id].R.set(R[id]);
        T[id].t = t[id];
    }
}

__global__ void pack_kernel(
    const Quaternion* R, 
    const Vector* t, // Vector3d / Vector3f
    Transform* T,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        T[id].R = R[id];
        T[id].t = t[id];
    }
}

__global__ void covParts_kernel(
    const Vector* a, 
    const Vector* b,
    Matrix3x3* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        C[id](0,0) = a[id].x * b[id].x;
        C[id](1,0) = a[id].x * b[id].y;
        C[id](2,0) = a[id].x * b[id].z;
        C[id](0,1) = a[id].y * b[id].x;
        C[id](1,1) = a[id].y * b[id].y;
        C[id](2,1) = a[id].y * b[id].z;
        C[id](0,2) = a[id].z * b[id].x;
        C[id](1,2) = a[id].z * b[id].y;
        C[id](2,2) = a[id].z * b[id].z;
    }
}

__global__ void covParts_kernel(
    const Vector* a, 
    const Vector* b,
    const bool* corr,
    Matrix3x3* C,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        if(corr[id])
        {
            C[id](0,0) = a[id].x * b[id].x;
            C[id](1,0) = a[id].x * b[id].y;
            C[id](2,0) = a[id].x * b[id].z;
            C[id](0,1) = a[id].y * b[id].x;
            C[id](1,1) = a[id].y * b[id].y;
            C[id](2,1) = a[id].y * b[id].z;
            C[id](0,2) = a[id].z * b[id].x;
            C[id](1,2) = a[id].z * b[id].y;
            C[id](2,2) = a[id].z * b[id].z;
        } else {
            C[id].setZeros();
        }
    }
}

template<unsigned int blockSize, typename T>
__device__ void warpReduce(volatile T* sdata, unsigned int tid)
{
    if(blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if(blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if(blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if(blockSize >=  8) sdata[tid] += sdata[tid + 4];
    if(blockSize >=  4) sdata[tid] += sdata[tid + 2];
    if(blockSize >=  2) sdata[tid] += sdata[tid + 1];
}



__global__ void normalizeInplace_kernel(
    Quaternion* q,
    unsigned int N)
{
    // TODO: this was empty. test this
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        q[id].normalizeInplace();
    }
}

template<unsigned int blockSize>
__global__ void cov_kernel(
    const Vector* v1,
    const Vector* v2,
    Matrix3x3* res,
    unsigned int N)
{
    __shared__ Matrix3x3 sdata[blockSize];
    
    const unsigned int tid = threadIdx.x;
    const unsigned int globId = N * blockIdx.x + threadIdx.x;
    const unsigned int rows = (N + blockSize - 1) / blockSize;

    sdata[tid].setZeros();
    for(unsigned int i=0; i<rows; i++)
    {
        if(tid + blockSize * i < N)
        {
            const Vector& a = v1[globId + blockSize * i];
            const Vector& b = v2[globId + blockSize * i];
            sdata[tid](0,0) += a.x * b.x;
            sdata[tid](1,0) += a.x * b.y;
            sdata[tid](2,0) += a.x * b.z;
            sdata[tid](0,1) += a.y * b.x;
            sdata[tid](1,1) += a.y * b.y;
            sdata[tid](2,1) += a.y * b.z;
            sdata[tid](0,2) += a.z * b.x;
            sdata[tid](1,2) += a.z * b.y;
            sdata[tid](2,2) += a.z * b.z;
        }
    }
    __syncthreads();

    for(unsigned int s = blockSize / 2; s > 32; s >>= 1)
    {
        if(tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if(tid < blockSize / 2 && tid < 32)
    {
        warpReduce<blockSize>(sdata, tid);
    }

    if(tid == 0)
    {
        res[blockIdx.x] = sdata[0] / static_cast<float>(N);
    }
}

} // namespace cuda

////////////
// #multNxN
void multNxN(
    const MemoryView<Quaternion, VRAM_CUDA>& A,
    const MemoryView<Quaternion, VRAM_CUDA>& B,
    MemoryView<Quaternion, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::multNxN_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Quaternion, VRAM_CUDA> multNxN(
    const MemoryView<Quaternion, VRAM_CUDA>& A, 
    const MemoryView<Quaternion, VRAM_CUDA>& B)
{
    Memory<Quaternion, VRAM_CUDA> C(A.size());
    // mult
    multNxN(A, B, C);
    return C;
}

void multNxN(
    const MemoryView<Quaternion, VRAM_CUDA>& A,
    const MemoryView<Vector, VRAM_CUDA>& b, 
    MemoryView<Vector, VRAM_CUDA>& c)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::multNxN_kernel<<<gridSize, blockSize>>>(A.raw(), b.raw(), c.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> multNxN(
    const MemoryView<Quaternion, VRAM_CUDA>& A,
    const MemoryView<Vector, VRAM_CUDA>& b)
{
    Memory<Vector, VRAM_CUDA> c(A.size());
    multNxN(A, b, c);
    return c;
}

/// TRANSFORM
void multNxN(
    const MemoryView<Transform, VRAM_CUDA>& T1,
    const MemoryView<Transform, VRAM_CUDA>& T2,
    MemoryView<Transform, VRAM_CUDA>& Tr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (T1.size() + blockSize - 1) / blockSize;
    cuda::multNxN_kernel<<<gridSize, blockSize>>>(T1.raw(), T2.raw(), Tr.raw(), T1.size());
    RM_CUDA_DEBUG();
}

Memory<Transform, VRAM_CUDA> multNxN(
    const MemoryView<Transform, VRAM_CUDA>& T1,
    const MemoryView<Transform, VRAM_CUDA>& T2)
{
    Memory<Transform, VRAM_CUDA> Tr(T1.size());
    multNxN(T1,T2,Tr);
    return Tr;
}

void multNxN(
    const MemoryView<Transform, VRAM_CUDA>& T,
    const MemoryView<Vector, VRAM_CUDA>& x,
    MemoryView<Vector, VRAM_CUDA>& c)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (T.size() + blockSize - 1) / blockSize;
    cuda::multNxN_kernel<<<gridSize, blockSize>>>(T.raw(), x.raw(), c.raw(), T.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> multNxN(
    const MemoryView<Transform, VRAM_CUDA>& T,
    const MemoryView<Vector, VRAM_CUDA>& x)
{
    Memory<Vector, VRAM_CUDA> c(T.size());
    multNxN(T,x,c);
    return c;
}

void multNxN(
    const MemoryView<Matrix3x3, VRAM_CUDA>& M1,
    const MemoryView<Matrix3x3, VRAM_CUDA>& M2,
    MemoryView<Matrix3x3, VRAM_CUDA>& Mr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M1.size() + blockSize - 1) / blockSize;
    cuda::multNxN_kernel<<<gridSize, blockSize>>>(M1.raw(), M2.raw(), Mr.raw(), M1.size());
    RM_CUDA_DEBUG();
}

Memory<Matrix3x3, VRAM_CUDA> multNxN(
    const MemoryView<Matrix3x3, VRAM_CUDA>& M1,
    const MemoryView<Matrix3x3, VRAM_CUDA>& M2)
{
    Memory<Matrix3x3, VRAM_CUDA> Mr(M1.size());
    multNxN(M1,M2,Mr);
    return Mr;
}

void multNxN(
    const MemoryView<Matrix3x3, VRAM_CUDA>& M1,
    const MemoryView<Matrix3x3, VRAM_CUDA>& M2,
    MemoryView<Quaternion, VRAM_CUDA>& Qres)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M1.size() + blockSize - 1) / blockSize;
    cuda::multNxN_conv_kernel<<<gridSize, blockSize>>>(M1.raw(), M2.raw(), Qres.raw(), M1.size());
    RM_CUDA_DEBUG();
}

void multNxN(
    const MemoryView<Matrix3x3, VRAM_CUDA>& M,
    const MemoryView<Vector, VRAM_CUDA>& x,
    MemoryView<Vector, VRAM_CUDA>& c)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M.size() + blockSize - 1) / blockSize;
    cuda::multNxN_kernel<<<gridSize, blockSize>>>(M.raw(), x.raw(), c.raw(), M.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> multNxN(
    const MemoryView<Matrix3x3, VRAM_CUDA>& M,
    const MemoryView<Vector, VRAM_CUDA>& x)
{
    Memory<Vector, VRAM_CUDA> c(M.size());
    multNxN(M, x, c);
    return c;
}

////////
// #multNx1
///
void multNx1(
    const MemoryView<Quaternion, VRAM_CUDA>& A,
    const MemoryView<Quaternion, VRAM_CUDA>& b,
    MemoryView<Quaternion, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::multNxN_kernel<<<gridSize, blockSize>>>(A.raw(), b.raw(), C.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Quaternion, VRAM_CUDA> multNx1(
    const MemoryView<Quaternion, VRAM_CUDA>& A, 
    const MemoryView<Quaternion, VRAM_CUDA>& b)
{
    Memory<Quaternion, VRAM_CUDA> C(A.size());
    // mult
    multNx1(A, b, C);
    return C;
}

void multNx1(
    const MemoryView<Quaternion, VRAM_CUDA>& A,
    const MemoryView<Vector, VRAM_CUDA>& b, 
    MemoryView<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::multNx1_kernel<<<gridSize, blockSize>>>(A.raw(), b.raw(), C.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> multNx1(
    const MemoryView<Quaternion, VRAM_CUDA>& A,
    const MemoryView<Vector, VRAM_CUDA>& b)
{
    Memory<Vector, VRAM_CUDA> C(A.size());
    multNx1(A, b, C);
    return C;
}

void multNx1(
    const MemoryView<Transform, VRAM_CUDA>& T1,
    const MemoryView<Transform, VRAM_CUDA>& t2,
    MemoryView<Transform, VRAM_CUDA>& Tr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (T1.size() + blockSize - 1) / blockSize;
    cuda::multNx1_kernel<<<gridSize, blockSize>>>(T1.raw(), t2.raw(), Tr.raw(), T1.size());
    RM_CUDA_DEBUG();
}

Memory<Transform, VRAM_CUDA> multNx1(
    const MemoryView<Transform, VRAM_CUDA>& T1,
    const MemoryView<Transform, VRAM_CUDA>& t2)
{
    Memory<Transform, VRAM_CUDA> Tr(T1.size());
    multNx1(T1,t2,Tr);
    return Tr;
}

void multNx1(
    const MemoryView<Transform, VRAM_CUDA>& T,
    const MemoryView<Vector, VRAM_CUDA>& x,
    MemoryView<Vector, VRAM_CUDA>& c)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (T.size() + blockSize - 1) / blockSize;
    cuda::multNx1_kernel<<<gridSize, blockSize>>>(T.raw(), x.raw(), c.raw(), T.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> multNx1(
    const MemoryView<Transform, VRAM_CUDA>& T,
    const MemoryView<Vector, VRAM_CUDA>& x)
{
    Memory<Vector, VRAM_CUDA> C(T.size());
    multNx1(T,x,C);
    return C;
}

void multNx1(
    const MemoryView<Matrix3x3, VRAM_CUDA>& M1,
    const MemoryView<Matrix3x3, VRAM_CUDA>& m2,
    MemoryView<Matrix3x3, VRAM_CUDA>& Mr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M1.size() + blockSize - 1) / blockSize;
    cuda::multNx1_kernel<<<gridSize, blockSize>>>(M1.raw(), m2.raw(), Mr.raw(), M1.size());
    RM_CUDA_DEBUG();
}

Memory<Matrix3x3, VRAM_CUDA> multNx1(
    const MemoryView<Matrix3x3, VRAM_CUDA>& M1,
    const MemoryView<Matrix3x3, VRAM_CUDA>& m2)
{
    Memory<Matrix3x3, VRAM_CUDA> Mr(M1.size());
    multNx1(M1,m2,Mr);
    return Mr;
}

void multNx1(
    const MemoryView<Matrix3x3, VRAM_CUDA>& M,
    const MemoryView<Vector, VRAM_CUDA>& x,
    MemoryView<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M.size() + blockSize - 1) / blockSize;
    cuda::multNx1_kernel<<<gridSize, blockSize>>>(M.raw(), x.raw(), C.raw(), M.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> multNx1(
    const MemoryView<Matrix3x3, VRAM_CUDA>& M,
    const MemoryView<Vector, VRAM_CUDA>& x)
{
    Memory<Vector, VRAM_CUDA> c(M.size());
    multNx1(M, x, c);
    return c;
}

void multNx1(
    const MemoryView<Matrix4x4, VRAM_CUDA>& M,
    const MemoryView<Vector, VRAM_CUDA>& x,
    MemoryView<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M.size() + blockSize - 1) / blockSize;
    cuda::multNx1_kernel<<<gridSize, blockSize>>>(M.raw(), x.raw(), C.raw(), M.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> multNx1(
    const MemoryView<Matrix4x4, VRAM_CUDA>& M,
    const MemoryView<Vector, VRAM_CUDA>& x)
{
    Memory<Vector, VRAM_CUDA> c(M.size());
    multNx1(M, x, c);
    return c;
}

/////////////
// #mult1xN
////////
void mult1xN(
    const MemoryView<Quaternion, VRAM_CUDA>& a,
    const MemoryView<Quaternion, VRAM_CUDA>& B,
    MemoryView<Quaternion, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (B.size() + blockSize - 1) / blockSize;
    cuda::mult1xN_kernel<<<gridSize, blockSize>>>(a.raw(), B.raw(), C.raw(), B.size());
    RM_CUDA_DEBUG();
}

Memory<Quaternion, VRAM_CUDA> mult1xN(
    const MemoryView<Quaternion, VRAM_CUDA>& a, 
    const MemoryView<Quaternion, VRAM_CUDA>& B)
{
    Memory<Quaternion, VRAM_CUDA> C(B.size());
    mult1xN(a, B, C);
    return C;
}

void mult1xN(
    const MemoryView<Quaternion, VRAM_CUDA>& a,
    const MemoryView<Vector, VRAM_CUDA>& B, 
    MemoryView<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (B.size() + blockSize - 1) / blockSize;
    cuda::mult1xN_kernel<<<gridSize, blockSize>>>(a.raw(), B.raw(), C.raw(), B.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> mult1xN(
    const MemoryView<Quaternion, VRAM_CUDA>& a,
    const MemoryView<Vector, VRAM_CUDA>& B)
{
    Memory<Vector, VRAM_CUDA> C(B.size());
    mult1xN(a, B, C);
    return C;
}

void mult1xN(
    const MemoryView<Transform, VRAM_CUDA>& t1,
    const MemoryView<Transform, VRAM_CUDA>& T2,
    MemoryView<Transform, VRAM_CUDA>& Tr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (T2.size() + blockSize - 1) / blockSize;
    cuda::mult1xN_kernel<<<gridSize, blockSize>>>(t1.raw(), T2.raw(), Tr.raw(), T2.size());
    RM_CUDA_DEBUG();
}

Memory<Transform, VRAM_CUDA> mult1xN(
    const MemoryView<Transform, VRAM_CUDA>& t1,
    const MemoryView<Transform, VRAM_CUDA>& T2)
{
    Memory<Transform, VRAM_CUDA> Tr(T2.size());
    mult1xN(t1, T2, Tr);
    return Tr;
}

void mult1xN(
    const MemoryView<Transform, VRAM_CUDA>& t,
    const MemoryView<Vector, VRAM_CUDA>& X,
    MemoryView<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (X.size() + blockSize - 1) / blockSize;
    cuda::mult1xN_kernel<<<gridSize, blockSize>>>(t.raw(), X.raw(), C.raw(), X.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> mult1xN(
    const MemoryView<Transform, VRAM_CUDA>& t,
    const MemoryView<Vector, VRAM_CUDA>& X)
{
    Memory<Vector, VRAM_CUDA> C(X.size());
    mult1xN(t, X, C);
    return C;
}

void mult1xN(
    const MemoryView<Matrix3x3, VRAM_CUDA>& m1,
    const MemoryView<Matrix3x3, VRAM_CUDA>& M2,
    MemoryView<Matrix3x3, VRAM_CUDA>& Mr)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (M2.size() + blockSize - 1) / blockSize;
    cuda::mult1xN_kernel<<<gridSize, blockSize>>>(m1.raw(), M2.raw(), Mr.raw(), M2.size());
    RM_CUDA_DEBUG();
}

Memory<Matrix3x3, VRAM_CUDA> mult1xN(
    const MemoryView<Matrix3x3, VRAM_CUDA>& m1,
    const MemoryView<Matrix3x3, VRAM_CUDA>& M2)
{
    Memory<Matrix3x3, VRAM_CUDA> Mr(M2.size());
    mult1xN(m1, M2, Mr);
    return Mr;
}

void mult1xN(
    const MemoryView<Matrix3x3, VRAM_CUDA>& m,
    const MemoryView<Vector, VRAM_CUDA>& X,
    MemoryView<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (X.size() + blockSize - 1) / blockSize;
    cuda::mult1xN_kernel<<<gridSize, blockSize>>>(m.raw(), X.raw(), C.raw(), X.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> mult1xN(
    const MemoryView<Matrix3x3, VRAM_CUDA>& m,
    const MemoryView<Vector, VRAM_CUDA>& X)
{
    Memory<Vector, VRAM_CUDA> C(X.size());
    mult1xN(m, X, C);
    return C;
}

void mult1xN(
    const MemoryView<Matrix4x4, VRAM_CUDA>& m,
    const MemoryView<Vector, VRAM_CUDA>& X,
    MemoryView<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (X.size() + blockSize - 1) / blockSize;
    cuda::mult1xN_kernel<<<gridSize, blockSize>>>(m.raw(), X.raw(), C.raw(), X.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> mult1xN(
    const MemoryView<Matrix4x4, VRAM_CUDA>& m,
    const MemoryView<Vector, VRAM_CUDA>& X)
{
    Memory<Vector, VRAM_CUDA> C(X.size());
    mult1xN(m, X, C);
    return C;
}

///////
// #add
void addNxN(
    const MemoryView<Vector, VRAM_CUDA>& A,
    const MemoryView<Vector, VRAM_CUDA>& B,
    MemoryView<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::addNxN_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> addNxN(
    const MemoryView<Vector, VRAM_CUDA>& A,
    const MemoryView<Vector, VRAM_CUDA>& B)
{
    Memory<Vector, VRAM_CUDA> C(A.size());
    addNxN(A, B, C);
    return C;
}

void addNxN(
    const MemoryView<float, VRAM_CUDA>& A,
    const MemoryView<float, VRAM_CUDA>& B,
    MemoryView<float, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::addNxN_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<float, VRAM_CUDA> addNxN(
    const MemoryView<float, VRAM_CUDA>& A,
    const MemoryView<float, VRAM_CUDA>& B)
{
    Memory<float, VRAM_CUDA> C(A.size());
    addNxN(A, B, C);
    return C;
}


////////
// #sub
void subNxN(
    const MemoryView<Vector, VRAM_CUDA>& A,
    const MemoryView<Vector, VRAM_CUDA>& B,
    MemoryView<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::subNxN_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> subNxN(
    const MemoryView<Vector, VRAM_CUDA>& A,
    const MemoryView<Vector, VRAM_CUDA>& B)
{
    Memory<Vector, VRAM_CUDA> C(A.size());
    subNxN(A, B, C);
    return C;
}

void subNx1(
    const MemoryView<Vector, VRAM_CUDA>& A,
    const MemoryView<Vector, VRAM_CUDA>& b,
    MemoryView<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::subNx1_kernel<<<gridSize, blockSize>>>(A.raw(), b.raw(), C.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> subNx1(
    const MemoryView<Vector, VRAM_CUDA>& A,
    const MemoryView<Vector, VRAM_CUDA>& b)
{
    Memory<Vector, VRAM_CUDA> C(A.size());
    subNx1(A, b, C);
    return C;
}

/////
// #transpose
void transpose(
    const MemoryView<Matrix3x3, VRAM_CUDA>& A, 
    MemoryView<Matrix3x3, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::transpose_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Matrix3x3, VRAM_CUDA> transpose(
    const MemoryView<Matrix3x3, VRAM_CUDA>& A)
{
    Memory<Matrix3x3, VRAM_CUDA> B(A.size());
    transpose(A, B);
    return B;
}

void transpose(
    const MemoryView<Matrix4x4, VRAM_CUDA>& A,
    MemoryView<Matrix4x4, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::transpose_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Matrix4x4, VRAM_CUDA> transpose(
    const MemoryView<Matrix4x4, VRAM_CUDA>& A)
{
    Memory<Matrix4x4, VRAM_CUDA> B(A.size());
    transpose(A, B);
    return B;
}

///////
// #transposeInplace
void transposeInplace(
    MemoryView<Matrix3x3, VRAM_CUDA>& A)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::transposeInplace_kernel<<<gridSize, blockSize>>>(A.raw(), A.size());
    RM_CUDA_DEBUG();
}

//////
// #invert
void invert(
    const MemoryView<Matrix3x3, VRAM_CUDA>& A, 
    MemoryView<Matrix3x3, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::invert_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Matrix3x3, VRAM_CUDA> invert(
    const MemoryView<Matrix3x3, VRAM_CUDA>& A)
{
    Memory<Matrix3x3, VRAM_CUDA> B(A.size());
    invert(A, B);
    return B;
}

void invert(
    const MemoryView<Matrix4x4, VRAM_CUDA>& A,
    MemoryView<Matrix4x4, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::invert_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Matrix4x4, VRAM_CUDA> invert(
    const MemoryView<Matrix4x4, VRAM_CUDA>& A)
{
    Memory<Matrix4x4, VRAM_CUDA> B(A.size());
    invert(A, B);
    return B;
}

void invert(
    const MemoryView<Transform, VRAM_CUDA>& A,
    MemoryView<Transform, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::invert_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Transform, VRAM_CUDA> invert(
    const MemoryView<Transform, VRAM_CUDA>& A)
{
    Memory<Transform, VRAM_CUDA> B(A.size());
    invert(A, B);
    return B;
}

//////
// #divNxN
void divNxN(
    const MemoryView<Vector, VRAM_CUDA>& A, 
    const MemoryView<unsigned int, VRAM_CUDA>& B,
    MemoryView<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::divNxN_conv_kernel<float><<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> divNxN(
    const MemoryView<Vector, VRAM_CUDA>& A, 
    const MemoryView<unsigned int, VRAM_CUDA>& B)
{
    Memory<Vector, VRAM_CUDA> C(A.size());
    divNxN(A, B, C);
    return C;
}

void divNxN(
    const MemoryView<Matrix3x3, VRAM_CUDA>& A, 
    const MemoryView<unsigned int, VRAM_CUDA>& B, 
    MemoryView<Matrix3x3, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::divNxN_conv_kernel<float><<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Matrix3x3, VRAM_CUDA> divNxN(
    const MemoryView<Matrix3x3, VRAM_CUDA>& A,
    const MemoryView<unsigned int, VRAM_CUDA>& B)
{
    Memory<Matrix3x3, VRAM_CUDA> C(A.size());
    divNxN(A, B, C);
    return C;
}

///////
// #divNxNIgnoreZeros
void divNxNIgnoreZeros(
    const MemoryView<Vector, VRAM_CUDA>& A, 
    const MemoryView<unsigned int, VRAM_CUDA>& B,
    MemoryView<Vector, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::divNxNIgnoreZeros_conv_kernel<float><<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> divNxNIgnoreZeros(
    const MemoryView<Vector, VRAM_CUDA>& A, 
    const MemoryView<unsigned int, VRAM_CUDA>& B)
{
    Memory<Vector, VRAM_CUDA> C(A.size());
    divNxNIgnoreZeros(A, B, C);
    return C;
}

void divNxNIgnoreZeros(
    const MemoryView<Matrix3x3, VRAM_CUDA>& A, 
    const MemoryView<unsigned int, VRAM_CUDA>& B,
    MemoryView<Matrix3x3, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::divNxNIgnoreZeros_conv_kernel<float><<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<Matrix3x3, VRAM_CUDA> divNxNIgnoreZeros(
    const MemoryView<Matrix3x3, VRAM_CUDA>& A, 
    const MemoryView<unsigned int, VRAM_CUDA>& B)
{
    Memory<Matrix3x3, VRAM_CUDA> C(A.size());
    divNxNIgnoreZeros(A, B, C);
    return C;
}

void divNxNIgnoreZeros(
    const MemoryView<float, VRAM_CUDA>& A, 
    const MemoryView<unsigned int, VRAM_CUDA>& B,
    MemoryView<float, VRAM_CUDA>& C)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::divNxNIgnoreZeros_conv_kernel<float><<<gridSize, blockSize>>>(A.raw(), B.raw(), C.raw(), A.size());
    RM_CUDA_DEBUG();
}

Memory<float, VRAM_CUDA> divNxNIgnoreZeros(
    const MemoryView<float, VRAM_CUDA>& A, 
    const MemoryView<unsigned int, VRAM_CUDA>& B)
{
    Memory<float, VRAM_CUDA> C(A.size());
    divNxNIgnoreZeros(A, B, C);
    return C;
}

////////
// #divNxNInplace
void divNxNInplace(
    MemoryView<Vector, VRAM_CUDA>& A, 
    const MemoryView<float, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::divNxNInplace_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
    RM_CUDA_DEBUG();
}

void divNxNInplace(
    MemoryView<Matrix3x3, VRAM_CUDA>& A, 
    const MemoryView<unsigned int, VRAM_CUDA>& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::divNxNInplace_kernel<<<gridSize, blockSize>>>(A.raw(), B.raw(), A.size());
}

////////
// #divNx1Inplace
void divNx1Inplace(
    MemoryView<Matrix3x3, VRAM_CUDA>& A, 
    const unsigned int& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::divNx1Inplace_kernel<<<gridSize, blockSize>>>(A.raw(), B, A.size());
    RM_CUDA_DEBUG();
}

void divNx1Inplace(
    MemoryView<Vector, VRAM_CUDA>& A, 
    const unsigned int& B)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (A.size() + blockSize - 1) / blockSize;
    cuda::divNx1Inplace_kernel<<<gridSize, blockSize>>>(A.raw(), B, A.size());
}


////////
// #convert
void convert(
    const MemoryView<uint8_t, VRAM_CUDA>& from, 
    MemoryView<float, VRAM_CUDA>& to)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (from.size() + blockSize - 1) / blockSize;
    cuda::convert_kernel<<<gridSize, blockSize>>>(from.raw(), to.raw(), from.size());
    RM_CUDA_DEBUG();
}

void convert(
    const MemoryView<bool, VRAM_CUDA>& from, 
    MemoryView<unsigned int, VRAM_CUDA>& to)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (from.size() + blockSize - 1) / blockSize;
    cuda::convert_kernel<<<gridSize, blockSize>>>(from.raw(), to.raw(), from.size());
}

void copy(const MemoryView<unsigned int, VRAM_CUDA>& from, 
    MemoryView<bool, VRAM_CUDA>& to)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (from.size() + blockSize - 1) / blockSize;
    cuda::convert_kernel<<<gridSize, blockSize>>>(from.raw(), to.raw(), from.size());
    RM_CUDA_DEBUG();
}

////////
// #pack
void pack(
    const MemoryView<Matrix3x3, VRAM_CUDA>& R,
    const MemoryView<Vector, VRAM_CUDA>& t,
    MemoryView<Transform, VRAM_CUDA>& T)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (R.size() + blockSize - 1) / blockSize;
    cuda::pack_kernel<<<gridSize, blockSize>>>(R.raw(), t.raw(), T.raw(), R.size());
}

void pack(
    const MemoryView<Quaternion, VRAM_CUDA>& R,
    const MemoryView<Vector, VRAM_CUDA>& t,
    MemoryView<Transform, VRAM_CUDA>& T)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (R.size() + blockSize - 1) / blockSize;
    cuda::pack_kernel<<<gridSize, blockSize>>>(R.raw(), t.raw(), T.raw(), R.size());
    RM_CUDA_DEBUG();
}

////////
// #multNxNTransposed
void multNxNTransposed(
    const MemoryView<Vector, VRAM_CUDA>& m1,
    const MemoryView<Vector, VRAM_CUDA>& m2,
    MemoryView<Matrix3x3, VRAM_CUDA>& Cs)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (m1.size() + blockSize - 1) / blockSize;
    cuda::covParts_kernel<<<gridSize, blockSize>>>(m1.raw(), m2.raw(), Cs.raw(), m1.size());
    RM_CUDA_DEBUG();
}

Memory<Matrix3x3, VRAM_CUDA> multNxNTransposed(
    const MemoryView<Vector, VRAM_CUDA>& m1,
    const MemoryView<Vector, VRAM_CUDA>& m2)
{
    Memory<Matrix3x3, VRAM_CUDA> Cs(m1.size());
    multNxNTransposed(m1, m2, Cs);
    return Cs;
}

void multNxNTransposed(
    const MemoryView<Vector, VRAM_CUDA>& m1,
    const MemoryView<Vector, VRAM_CUDA>& m2,
    const MemoryView<bool, VRAM_CUDA>& mask,
    MemoryView<Matrix3x3, VRAM_CUDA>& Cs)
{
    constexpr unsigned int blockSize = 64;
    const unsigned int gridSize = (m1.size() + blockSize - 1) / blockSize;
    cuda::covParts_kernel<<<gridSize, blockSize>>>(m1.raw(), m2.raw(), mask.raw(), Cs.raw(), m1.size());
    RM_CUDA_DEBUG();
}
    
Memory<Matrix3x3, VRAM_CUDA> multNxNTransposed(
    const MemoryView<Vector, VRAM_CUDA>& m1,
    const MemoryView<Vector, VRAM_CUDA>& m2,
    const MemoryView<bool, VRAM_CUDA>& mask)
{
    Memory<Matrix3x3, VRAM_CUDA> Cs(m1.size());
    multNxNTransposed(m1, m2, mask, Cs);
    return Cs;
}

///////
// #normalize
void normalizeInplace(MemoryView<Quaternion, VRAM_CUDA>& q)
{
    constexpr unsigned int blockSize = 1024;
    const unsigned int gridSize = (q.size() + blockSize - 1) / blockSize;
    cuda::normalizeInplace_kernel<<<gridSize, blockSize>>>(q.raw(), q.size());
    RM_CUDA_DEBUG();
}

///////
// #setter

namespace cuda
{

template<typename T>
__global__ void setIdentity_kernel(
    T* data,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        data[id] = T::Identity();
    }
}

} // namespace cuda

void setIdentity(MemoryView<Quaternion, VRAM_CUDA>& qs)
{
    constexpr unsigned int blockSize = 1024;
    const unsigned int gridSize = (qs.size() + blockSize - 1) / blockSize;
    cuda::setIdentity_kernel<<<gridSize, blockSize>>>(qs.raw(), qs.size());
    RM_CUDA_DEBUG();
}

void setIdentity(MemoryView<Transform, VRAM_CUDA>& Ts)
{
    constexpr unsigned int blockSize = 1024;
    const unsigned int gridSize = (Ts.size() + blockSize - 1) / blockSize;
    cuda::setIdentity_kernel<<<gridSize, blockSize>>>(Ts.raw(), Ts.size());
    RM_CUDA_DEBUG();
}

void setIdentity(MemoryView<Matrix3x3, VRAM_CUDA>& Ms)
{
    constexpr unsigned int blockSize = 1024;
    const unsigned int gridSize = (Ms.size() + blockSize - 1) / blockSize;
    cuda::setIdentity_kernel<<<gridSize, blockSize>>>(Ms.raw(), Ms.size());
    RM_CUDA_DEBUG();
}

void setIdentity(MemoryView<Matrix4x4, VRAM_CUDA>& Ms)
{
    constexpr unsigned int blockSize = 1024;
    const unsigned int gridSize = (Ms.size() + blockSize - 1) / blockSize;
    cuda::setIdentity_kernel<<<gridSize, blockSize>>>(Ms.raw(), Ms.size());
    RM_CUDA_DEBUG();
}

void setZeros(MemoryView<Matrix3x3, VRAM_CUDA>& Ms)
{
    hipMemset(Ms.raw(), 0, Ms.size() * sizeof(Matrix3x3) );
}

void setZeros(MemoryView<Matrix4x4, VRAM_CUDA>& Ms)
{
    hipMemset(Ms.raw(), 0, Ms.size() * sizeof(Matrix4x4) );
}

namespace cuda
{
//////////
// #sum
// TODO: check perfomance of sum_kernel
template<unsigned int nMemElems, typename T>
__global__ void sum_kernel(
    const T* data,
    T* res,
    unsigned int N)
{
    // sharedMemElements per block

    // Many blocks stategy
    // rows=2
    // 
    //   blockId=0                  blockId=1
    // sharedMemElements |  -- sharedMemElements --- 
    // [ 1,  3,  5,  7]    [9,  11, 13, 15]
    // [ 2,  4,  6,  8]    [10, 12, 14, 16]
    //   |   |   |   |       |   |   |   | 
    // [ 3,  7, 11, 15]    [19, 23, 27, 31]
    // [10, 26]            [42, 58]
    // [36]                [100]
    __shared__ T sdata[nMemElems];

    const unsigned int n_threads = blockDim.x;
    const unsigned int n_blocks = gridDim.x;

    const unsigned int total_threads = n_threads * n_blocks;
    const unsigned int n_rows = (N + total_threads - 1) / total_threads;
    const unsigned int n_elems_per_block = n_rows * nMemElems;

    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;
    const unsigned int glob_shift = n_elems_per_block * bid;

    sdata[tid] *= 0.0;
    for(unsigned int i=0; i<n_rows; i++)
    {
        const unsigned int data_id = glob_shift + i * nMemElems + tid; // advance one row
        if(data_id < N)
        {
            sdata[tid] += data[data_id];
        }
    }
    __syncthreads();

    for(unsigned int s = nMemElems / 2; s > 32; s >>= 1)
    {
        if(tid < s)
        {
            sdata[tid] += sdata[tid + s];
        } else {
            // TODO: can this thread do something useful in the meantime?
        }
        __syncthreads();
    }

    if(tid < nMemElems / 2 && tid < 32)
    {
        warpReduce<nMemElems>(sdata, tid);
    }

    // Do this instead for types that have no volotile operators implemented:
    // for(unsigned int s = nMemElems / 2; s > 0; s >>= 1)
    // {
    //     if(tid < s)
    //     {
    //         sdata[tid] += sdata[tid + s];
    //     } else {
    //         // TODO: can this thread do something useful in the meantime?
    //     }
    //     __syncthreads();
    // }
    // the warpReduce gives a comparable small performance boost with my profiling tests
    // so, this would be still a good we to do a reduction

    if(tid == 0)
    {
        res[bid] = sdata[0];
    }
}

} // namespace cuda

void sum(
    const MemoryView<Vector, VRAM_CUDA>& data,
    MemoryView<Vector, VRAM_CUDA>& s)
{
    const unsigned int n_outputs = s.size(); // also number of blocks
    constexpr unsigned int n_threads = 1024; // also shared mem
    // the rest is computed automatically

    cuda::sum_kernel<n_threads> <<<n_outputs, n_threads>>>(data.raw(), s.raw(), data.size());
    RM_CUDA_DEBUG();
}

Memory<Vector, VRAM_CUDA> sum(
    const MemoryView<Vector, VRAM_CUDA>& data)
{
    Memory<Vector, VRAM_CUDA> s(1);
    sum(data, s);
    return s;
}

void sum(
    const MemoryView<int, VRAM_CUDA>& data,
    MemoryView<int, VRAM_CUDA>& s)
{
    const unsigned int n_outputs = s.size(); // also number of blocks
    constexpr unsigned int n_threads = 1024; // also shared mem

    cuda::sum_kernel<n_threads> <<<n_outputs, n_threads>>>(data.raw(), s.raw(), data.size());
    RM_CUDA_DEBUG();
}

Memory<int, VRAM_CUDA> sum(
    const MemoryView<int, VRAM_CUDA>& data)
{
    Memory<int, VRAM_CUDA> s(1);
    sum(data, s);
    return s;
}

//////////
// #mean
void mean(
    const MemoryView<Vector, VRAM_CUDA>& X,
    MemoryView<Vector, VRAM_CUDA>& res)
{
    sum(X, res);
    divNx1Inplace(res, X.size());
}

Memory<Vector, VRAM_CUDA> mean(
    const MemoryView<Vector, VRAM_CUDA>& X)
{
    Memory<Vector, VRAM_CUDA> res(1);
    mean(X, res);
    return res;
}

//////////
// #cov
void cov(
    const MemoryView<Vector, VRAM_CUDA>& v1,
    const MemoryView<Vector, VRAM_CUDA>& v2,
    MemoryView<Matrix3x3, VRAM_CUDA>& C)
{
    cuda::cov_kernel<1024> <<<1, 1024>>>(v1.raw(), v2.raw(), C.raw(), v1.size());
    RM_CUDA_DEBUG();
}

Memory<Matrix3x3, VRAM_CUDA> cov(
    const MemoryView<Vector, VRAM_CUDA>& v1,
    const MemoryView<Vector, VRAM_CUDA>& v2
)
{
    Memory<Matrix3x3, VRAM_CUDA> C(1);
    cov(v1, v2, C);
    return C;
}

namespace cuda
{

__global__ void svd_kernel(
    const Matrix3x3* As,
    Matrix3x3* Us,
    Matrix3x3* Ws,
    Matrix3x3* Vs, 
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        svd(As[id], Us[id], Ws[id], Vs[id]);
    }
}

} // namespace cuda

void svd(
    const MemoryView<Matrix3x3, VRAM_CUDA>& As,
    MemoryView<Matrix3x3, VRAM_CUDA>& Us,
    MemoryView<Matrix3x3, VRAM_CUDA>& Ws,
    MemoryView<Matrix3x3, VRAM_CUDA>& Vs
)
{
    constexpr unsigned int blockSize = 512;
    const unsigned int gridSize = (As.size() + blockSize - 1) / blockSize;
    cuda::svd_kernel<<<gridSize, blockSize>>>(As.raw(), Us.raw(), Ws.raw(), Vs.raw(), As.size());
    RM_CUDA_DEBUG();
}

namespace cuda
{

__global__ void svd_kernel(
    const Matrix3x3* As,
    Matrix3x3* Us,
    Vector3* ws,
    Matrix3x3* Vs,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        svd(As[id], Us[id], ws[id], Vs[id]);
    }
}

} // namespace cuda

void svd(
    const MemoryView<Matrix3x3, VRAM_CUDA>& As,
    MemoryView<Matrix3x3, VRAM_CUDA>& Us,
    MemoryView<Vector3, VRAM_CUDA>& ws,
    MemoryView<Matrix3x3, VRAM_CUDA>& Vs
)
{
    constexpr unsigned int blockSize = 512;
    const unsigned int gridSize = (As.size() + blockSize - 1) / blockSize;
    cuda::svd_kernel<<<gridSize, blockSize>>>(As.raw(), Us.raw(), ws.raw(), Vs.raw(), As.size());
    RM_CUDA_DEBUG();
}

namespace cuda
{
__global__ void umeyama_transform_kernel(
    Transform* Ts,
    const Vector3* ds,
    const Vector3* ms,
    const Matrix3x3* Cs,
    const unsigned int* n_meas,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        Ts[id] = umeyama_transform(ds[id], ms[id], Cs[id], n_meas[id]);
    }
}
} // namespace cuda

void umeyama_transform(
    MemoryView<Transform, VRAM_CUDA>& Ts,
    const MemoryView<Vector3, VRAM_CUDA>& ds,
    const MemoryView<Vector3, VRAM_CUDA>& ms,
    const MemoryView<Matrix3x3, VRAM_CUDA>& Cs,
    const MemoryView<unsigned int, VRAM_CUDA>& n_meas)
{
    constexpr unsigned int blockSize = 256;
    const unsigned int gridSize = (Ts.size() + blockSize - 1) / blockSize;
    cuda::umeyama_transform_kernel<<<gridSize, blockSize>>>(Ts.raw(), ds.raw(), ms.raw(), Cs.raw(), n_meas.raw(), Ts.size());
    RM_CUDA_DEBUG();
}

Memory<Transform, VRAM_CUDA> umeyama_transform(
    const MemoryView<Vector3, VRAM_CUDA>& ds,
    const MemoryView<Vector3, VRAM_CUDA>& ms,
    const MemoryView<Matrix3x3, VRAM_CUDA>& Cs,
    const MemoryView<unsigned int, VRAM_CUDA>& n_meas)
{
    Memory<Transform, VRAM_CUDA> ret(ds.size());
    umeyama_transform(ret, ds, ms, Cs, n_meas);
    return ret;
}

namespace cuda
{
__global__ void umeyama_transform_kernel(
    Transform* Ts,
    const Vector3* ds,
    const Vector3* ms,
    const Matrix3x3* Cs,
    unsigned int N)
{
    const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
    {
        Ts[id] = umeyama_transform(ds[id], ms[id], Cs[id]);
    }
}
} // namespace cuda

void umeyama_transform(
    MemoryView<Transform, VRAM_CUDA>& Ts,
    const MemoryView<Vector3, VRAM_CUDA>& ds,
    const MemoryView<Vector3, VRAM_CUDA>& ms,
    const MemoryView<Matrix3x3, VRAM_CUDA>& Cs)
{
    constexpr unsigned int blockSize = 256;
    const unsigned int gridSize = (Ts.size() + blockSize - 1) / blockSize;
    cuda::umeyama_transform_kernel<<<gridSize, blockSize>>>(Ts.raw(), ds.raw(), ms.raw(), Cs.raw(), Ts.size());
    RM_CUDA_DEBUG();
}

Memory<Transform, VRAM_CUDA> umeyama_transform(
    const MemoryView<Vector3, VRAM_CUDA>& ds,
    const MemoryView<Vector3, VRAM_CUDA>& ms,
    const MemoryView<Matrix3x3, VRAM_CUDA>& Cs)
{
    Memory<Transform, VRAM_CUDA> ret(ds.size());
    umeyama_transform(ret, ds, ms, Cs);
    return ret;
}

namespace cuda
{

template<unsigned int nMemElems>
__global__ void print_variables(
    const int* data,
    int* res,
    unsigned int N)
{
    const unsigned int n_threads_per_block = blockDim.x;
    const unsigned int n_blocks = gridDim.x;

    const unsigned int n_threads_total = n_threads_per_block * n_blocks;
    const unsigned int n_rows_per_block = (N + n_threads_total - 1) / n_threads_total;
    const unsigned int n_elems_per_block = n_rows_per_block * nMemElems;

    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;
    const unsigned int block_shift = n_elems_per_block * bid;

    if(tid == 0 && bid == 0)
    {
        printf("Variables:\n");
        printf("- # blocks, threads: %u, %u\n", n_blocks, n_threads_per_block);
        printf("- # rows: %u\n", n_rows_per_block);
        printf("- # elems per block: %u\n", n_elems_per_block);
        printf("- block shift: %u\n", block_shift);
    }
}

//////////
// #sum
// TODO: check perfomance of sum_kernel
template<unsigned int nMemElems>
__global__ void sum_kernel_test(
    const int* data,
    int* res,
    unsigned int N)
{
    // sharedMemElements per block

    // Many blocks stategy
    // rows=2
    // 
    //   blockId=0                  blockId=1
    // sharedMemElements |  -- sharedMemElements --- 
    // [ 1,  3,  5,  7]    [9,  11, 13, 15]
    // [ 2,  4,  6,  8]    [10, 12, 14, 16]
    //   |   |   |   |       |   |   |   | 
    // [ 3,  7, 11, 15]    [19, 23, 27, 31]
    // [10, 26]            [42, 58]
    // [36]                [100]
    __shared__ int sdata[nMemElems];

    const unsigned int n_threads_per_block = blockDim.x;
    const unsigned int n_blocks = gridDim.x;

    const unsigned int n_threads_total = n_threads_per_block * n_blocks;
    const unsigned int n_rows_per_block = (N + n_threads_total - 1) / n_threads_total;
    const unsigned int n_elems_per_block = n_rows_per_block * nMemElems;

    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;
    const unsigned int block_shift = n_elems_per_block * bid;

    sdata[tid] = 0;
    for(unsigned int i=0; i<n_rows_per_block; i++)
    {
        const unsigned int data_id = block_shift + i * nMemElems + tid; // advance one row
        if(data_id < N)
        {
            printf("b%u, t%u: data -> smem: %u -> %u\n", bid, tid, data_id, tid);
            sdata[tid] += data[data_id];
        }
    }
    __syncthreads();

    unsigned int depth = 0;
    for(unsigned int s = nMemElems / 2; s > 0; s >>= 1)
    {
        if(tid < s)
        {
            printf("b%u, t%u: smem reduce: (%u + %u)_%u -> (%u)_%u\n", bid, tid, tid, tid + s, depth, tid, depth+1);
            sdata[tid] += sdata[tid + s];
        } else {
            // TODO: can this thread do something useful in the meantime?
        }
        depth++;
        __syncthreads();
    }

    if(tid == 0)
    {
        res[bid] = sdata[0];
    }
}

} // namespace cuda

void sum_reduce_test_t1(
    const MemoryView<int, VRAM_CUDA>& data, 
    MemoryView<int, VRAM_CUDA> results)
{
    const unsigned int n_outputs = results.size(); // also number of blocks
    constexpr unsigned int n_threads = 1; // also shared mem
    cuda::print_variables<n_threads> <<<n_outputs, n_threads>>>(data.raw(), results.raw(), data.size());
    cuda::sum_kernel_test<n_threads> <<<n_outputs, n_threads>>>(data.raw(), results.raw(), data.size());
}

void sum_reduce_test_t2(
    const MemoryView<int, VRAM_CUDA>& data, 
    MemoryView<int, VRAM_CUDA> results)
{
    const unsigned int n_outputs = results.size(); // also number of blocks
    constexpr unsigned int n_threads = 2; // also shared mem
    cuda::print_variables<n_threads> <<<n_outputs, n_threads>>>(data.raw(), results.raw(), data.size());
    cuda::sum_kernel_test<n_threads> <<<n_outputs, n_threads>>>(data.raw(), results.raw(), data.size());
}

void sum_reduce_test_t4(
    const MemoryView<int, VRAM_CUDA>& data, 
    MemoryView<int, VRAM_CUDA> results)
{
    const unsigned int n_outputs = results.size(); // also number of blocks
    constexpr unsigned int n_threads = 4; // also shared mem
    cuda::print_variables<n_threads> <<<n_outputs, n_threads>>>(data.raw(), results.raw(), data.size());
    cuda::sum_kernel_test<n_threads> <<<n_outputs, n_threads>>>(data.raw(), results.raw(), data.size());
}

void sum_reduce_test_t8(
    const MemoryView<int, VRAM_CUDA>& data, 
    MemoryView<int, VRAM_CUDA> results)
{
    const unsigned int n_outputs = results.size(); // also number of blocks
    constexpr unsigned int n_threads = 8; // also shared mem
    cuda::print_variables<n_threads> <<<n_outputs, n_threads>>>(data.raw(), results.raw(), data.size());
    cuda::sum_kernel_test<n_threads> <<<n_outputs, n_threads>>>(data.raw(), results.raw(), data.size());
}

} // namespace rmagine