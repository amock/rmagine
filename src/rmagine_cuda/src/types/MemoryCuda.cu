#include "rmagine/types/MemoryCuda.hpp"
#include "rmagine/util/cuda/CudaStream.hpp"

#include <hip/hip_runtime.h>
#include "rmagine/util/cuda/CudaDebug.hpp"


namespace rmagine {

// CUDA HELPER
namespace cuda {

void* memcpyHostToDevice(void* dest, const void* src, size_t count)
{
    RM_CUDA_CHECK( hipMemcpy(dest, src, count, hipMemcpyHostToDevice) );
    return dest;
}

void* memcpyHostToDevice(void* dest, const void* src, size_t count, CudaStreamPtr stream)
{
    RM_CUDA_CHECK( hipMemcpyAsync(dest, src, count, hipMemcpyHostToDevice, stream->handle()) );
    return dest;
}

void* memcpyDeviceToHost(void* dest, const void* src, size_t count)
{
    RM_CUDA_CHECK( hipMemcpy(dest, src, count, hipMemcpyDeviceToHost) );
    return dest;
}

void* memcpyDeviceToHost(void* dest, const void* src, size_t count, CudaStreamPtr stream)
{
    RM_CUDA_CHECK( hipMemcpyAsync(dest, src, count, hipMemcpyDeviceToHost, stream->handle()) );
    return dest;
}

void* memcpyDeviceToDevice(void* dest, const void* src, size_t count)
{
    RM_CUDA_CHECK( hipMemcpy(dest, src, count, hipMemcpyDeviceToDevice) );
    return dest;
}

void* memcpyDeviceToDevice( void* dest, const void* src, size_t count, CudaStreamPtr stream)
{
    RM_CUDA_CHECK( hipMemcpyAsync(dest, src, count, hipMemcpyDeviceToDevice, stream->handle()) );
    return dest;
}

void* memcpyHostToHost(void* dest, const void* src, size_t count)
{
    RM_CUDA_CHECK( hipMemcpy(dest, src, count, hipMemcpyHostToHost) );
    return dest;
}

void* memcpyHostToHost(     void* dest, const void* src, size_t count, CudaStreamPtr stream)
{
    RM_CUDA_CHECK( hipMemcpyAsync(dest, src, count, hipMemcpyHostToHost, stream->handle()) );
    return dest;
}

void** malloc(void** ptr, size_t count)
{
    RM_CUDA_CHECK( hipMalloc(ptr, count) );
    return ptr;
}

void** mallocHost(void** ptr, size_t count)
{
    RM_CUDA_CHECK( hipHostMalloc(ptr, count) );
    return ptr;
}

void** mallocManaged(void** ptr, size_t count)
{
    RM_CUDA_CHECK( hipMallocManaged(ptr, count) );
    return ptr;
}

void* free(void* ptr)
{
    RM_CUDA_CHECK( hipFree(ptr) );
    return ptr;
}

void* freeHost(void* ptr)
{
    RM_CUDA_CHECK( hipHostFree(ptr) );
    return ptr;
}


} // namespace cuda

} // namespace mamcl